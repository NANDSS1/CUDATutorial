#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#define THREAD_PER_BLOCK 256

/*进行了一个完全for展开的操作，但是我不太清楚什么会快了？*/
/*只要记住一点就行了，for展开肯定比单独写for好*/

/*具体来说，通过条件分支和循环展开，使得在每个条件分支内的指令能够在编译时确定。*/
// latency: 0.656ms
template <int blockSize>
__device__ void BlockSharedMemReduce(float* smem) {
  if (blockSize >= 1024) {
    if (threadIdx.x < 512) {
      smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
  }
  if (blockSize >= 512) {
    if (threadIdx.x < 256) {
      smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (threadIdx.x < 128) {
      smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (threadIdx.x < 64) {
      smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();
  }
  // the final warp
  if (threadIdx.x < 32) {//对前64个数据进行规约
    volatile float* vshm = smem;//定义一组寄存器
    if (blockDim.x >= 64) {
      vshm[threadIdx.x] += vshm[threadIdx.x + 32];
    }
    vshm[threadIdx.x] += vshm[threadIdx.x + 16];
    vshm[threadIdx.x] += vshm[threadIdx.x + 8];
    vshm[threadIdx.x] += vshm[threadIdx.x + 4];
    vshm[threadIdx.x] += vshm[threadIdx.x + 2]; 
    vshm[threadIdx.x] += vshm[threadIdx.x + 1];
  }
}

template <int blockSize>
__global__ void reduce_v5(float *d_in, float *d_out){
    __shared__ float smem[THREAD_PER_BLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    // load: 每个线程加载两个元素到shared mem对应位置
    smem[tid] = d_in[i] + d_in[i + blockDim.x];
    __syncthreads();
    // compute: reduce in shared mem
    BlockSharedMemReduce<blockSize>(smem);

    // store: write back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v5<blockSize / 2><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("resPerBlock : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v5 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
