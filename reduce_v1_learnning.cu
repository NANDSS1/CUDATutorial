#include<bits/stdc++.h>
#include <hip/hip_runtime.h>


template<int blockSize>//模板，接收一个整形数据，执行的时候填上这个整形数据reduce_kernel<256><<<gridSize, blockSize>>>(d_input, d_output, N);

__global__ void reduce_v0(float* d_in,float* d_out){
    __shared__ float smem[blockSize];//申请这么大的共享内存在一个block里面，属于是block,它用于在线程块内部存储临时数据
    int tid = threadIdx.x;//获取线程在block里面的id
    int gtid = blockIdx.x * blockSize + threadIdx.x;//获取一个全局id

    smem[tid] = d_in[gtid];//把数据依次加载到block的shared memory里面
    __syncthreads();//等待线程同步

    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            smem[index] += smem[index + s];
            //blockDim.x = 256
            //2*s*tid < 256
            //s*tid < 128

            //第一轮 s = 1 tid<128 前四个warp所有线程都能同时运行分支，不会发生warp divergence
            //第二轮 s = 2 tid<64 前两个warp所有线程都能同时运行分支，不会发生warp divergence
            //第三轮 s = 4 tid<32 前一个warp所有线程都能同时运行一个分支，不会发生warp divergence

            //第四轮 s = 8 tid<16 第一个warp只有前16个线程能同时运行一个分支，发生了warp divergence
        }
        __syncthreads();
    }

    // store: write back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
        //把每个block的tid0的结果写会global memory
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    hipSetDevice(0);//选择第一个gpu
    hipDeviceProp_t deviceProp;//存储关于 GPU 设备的属性信息
    hipGetDeviceProperties(&deviceProp, 0);//获取第一个gpu的属性
    const int blockSize = 256;//设定block size
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //(N + 256 - 1) / 256表示对N/256的向上取整，保证所有数据都能分配上thread
    //-1的原因是如果N刚好是256的倍数，分配的block也不会多，如果N=512，不-1，分配的block就是3，如果-1，分配的block就是2

    float *a = (float *)malloc(N * sizeof(float));//申请host上的内存
    float *d_a;
    hipMalloc((void**)&d_a,N * sizeof(float));//申请device上的内存

    float *out = (float*)malloc((GridSize) * sizeof(float));//申请的float数量和block数量一致
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }//初始化host上的数据


    float groudtruth = N * 1.0f;//累积误差的标准

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);//host cpoy data to device

    dim3 Grid(GridSize);//在x方向申请这么多个block，用作<<<,>>>的第一个参数
    dim3 Block(blockSize);//在x方向申请这么多个thread，用作<<<,>>>的第二个参数

    hipEvent_t start, stop;//定义两个事件用作定时
    hipEventCreate(&start);//创建一个开始定时器
    hipEventCreate(&stop);//创建一个结束定时器
    hipEventRecord(start);//开始定时器记录一个时间戳

    reduce_v0<blockSize><<<Grid,Block>>>(d_a, d_out);//执行核函数,这里用模板传参的

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);//device copy data to host

    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
        if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        //for(int i = 0; i < GridSize;i++){
            //printf("res per block : %lf ",out[i]);
        //}
        //printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v0 latency = %f ms\n", milliseconds);


    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);



}